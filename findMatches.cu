#include "hip/hip_runtime.h"
/*  ###########################DESCRIPTION###############################
 *  Written by XXXXXXX (University of XXXXXX) as a part of a bachelor's 
 *  thesis that uses a blockmatching algorithm  to gather a statistical 
 *  population for denoising single pixels in an image.
 */

/*  This function contains the findMatches CUDA kernel that will find 
 *  matches for every pixel in an image of size M by N (rows x columns).
 *  The algorithm is based on the following paper:
 *  http://www.mia.uni-saarland.de/Publications/zimmer-lnla08.pdf
 */



 /* ###########################STYLE NOTES###############################
  * Device variables will have the prefix d_, host variables h_
  * The suffix _ptr will be used to denote that the variable is a pointer.
  *
  * Throughout this code, I will insert footnotes inside comments of the 
  * format (#) which -unsurprisingly- can be found at the bottom. This is 
  * first of all to keep the code compact, but also to allow both the 
  * reader (you) as the developer (me) to understand this code, i.e. "why 
  * use datatype X", "why do loop Y like this", "why is thisindex Z minus 
  * one", ...  
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <math.h>

/*  Kernel code, using floats because doubles can drastically hurt perfor-
 *  mance.
 */
void __global__ findMatches(const float* d_img, const int M, const int N){
    //Array coordinates of the reference block. 
    const int i = blockDim.x*blockIdx.x+threadIdx.x;
    const int j = blockDim.y*blockIdx.y+threadIdx.y;
    if (i < M && j < N){
        // Do stuff...
    }
}



// Call in matlab like this:
//[plhs[0],plhs[1],plhs[...],plhs[nrhs-1]]=filename(prhs[0],prhs[1],prhs[...],prhs[nrhs-1])
void mexFunction(   int nlhs, mxArray *plhs[],
                    int nrhs, mxArray const *prhs[]){
    /* prhs argument explanation:
     *plhs[0]: mxGPUarray that contains the image. (1)
     */
    
    //Variable declarations
    
    //Initialize MathWorks GPU API. 
    mxInitGPU();
     
    //Kernel parameters
	/*Figure out grid layout. We'll use a 2D grid where each thread corresponds with
       *one pixel. We'll go for 1024 threads per block, which for a 2.1 CC device gives
       *us 67% occupancy.
      */
	hipDeviceProp_t device;
	hipGetDeviceProperties(&device,0);
	const int MaxThreadsPerBlock=device.maxThreadsPerBlock;
	dim3 BlockDim;
	BlockDim.x=sqrt((double) MaxThreadsPerBlock);
      BlockDim.y=sqrt((double) MaxThreadsPerBlock);
      mexPrintf("\n x: %u y:%u \n",BlockDim.x,BlockDim.y);
}



/*####################### FOOTNOTES ##################################
 *(1)	I could also accept a (host) mxArray, but this would lengthen the 
 *      code with all sorts of ugly CUDA API calls that do the same thing 
 *      as mxGpuArray. 
 *       
 */
